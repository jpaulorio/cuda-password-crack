#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/stat.h>
#include <dirent.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define total_no_ascii_chars 95
#define max_encrypted_pwd_length 8

extern "C"
void ulong_to_char_array(unsigned long search_pos, char *output);

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

__device__ int d_strcmp (char *s1, char *s2, uint size) {
    for(int i=0; i < size; i++) {
        if(s1[i] != s2[i])
            return 1;
    }
    return 0;
}

__device__ void d_strcpy (char *origin, char *destination, uint size) {
    for (int i=0; i < size; i++) {
        destination[i] = origin[i];
    }
}

__device__ unsigned long d_pow(unsigned long n, unsigned long power) {
    unsigned long result = 1;
    for (int i=0; i<power; i++) {
        result = result * n;
    }
    return result;
}

__device__ unsigned long d_char_array_to_ulong(char *input, uint array_lenght) {
    unsigned long result = 0;
    for (int i=0; i < array_lenght && input[i] != 0; i++) {
        result += (input[i] - 32) * d_pow(total_no_ascii_chars, i);
    }
    return result;
}

__device__ unsigned long d_encrypt(unsigned long input, uint encryption_key) {
    unsigned long tmp_pwd = input * encryption_key;
    return tmp_pwd;
}

__device__ int g_found = 0;
__device__ unsigned long d_answer = 0;

__global__ void
crackPassword( char *g_encrypted_password, unsigned long pageDim, unsigned long pageId)
{
    char encrypted_password[max_encrypted_pwd_length];
    
    const unsigned long tidx = threadIdx.x;
    const unsigned long tidy = threadIdx.y;
    const unsigned long bid = blockIdx.x;
    const unsigned int num_threads = blockDim.x;
    const unsigned long global_tid = (pageId * gridDim.x * blockDim.x) + (bid * num_threads) + tidx;

    const uint encryption_keys[] = {
        31, 37, 41, 43, 47, 53, 59, 61, 67, 71,
        73, 79, 83, 89, 97, 101, 103, 107, 109, 113,
        127, 131, 137, 139, 149, 151, 157, 163, 167, 173,
        179, 181, 191, 193, 197, 199, 211, 223, 227, 229,
        233, 239, 241, 251, 257, 263, 269, 271, 277, 281,
        283, 293, 307, 311, 313, 317, 331, 337, 347, 349,
        353, 359, 367, 373, 379, 383, 389, 397, 401, 409,
        419, 421, 431, 433, 439, 443, 449, 457, 461, 463, 
        467, 479, 487, 491, 499, 503, 509, 521, 523, 541
    };
    uint key = encryption_keys[tidy];

    d_strcpy(g_encrypted_password, encrypted_password, 7);

    if (g_found) {
        return;
    }

    unsigned long long_encrypted = d_char_array_to_ulong(encrypted_password, 7);
    unsigned long tmp_encrypted = d_encrypt(global_tid, key);

    if (long_encrypted == tmp_encrypted) {
        d_answer = global_tid;
        g_found = 1;
    }
}

int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

void
runTest(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    const unsigned int pwd_max_size = 32 + 1;
    const uint key_list_size = 90;
    
    char encrypted_password[pwd_max_size];
    printf("Enter the encrypted password:\n");
    scanf("%7s", encrypted_password);
    
    uint pwd_size = strlen(encrypted_password);
    
    unsigned int pwd_mem_size = (pwd_size + 1) * sizeof(char);
    unsigned long search_space_size = pow(total_no_ascii_chars, 5);
    printf("Search space size: %lu\n", search_space_size * key_list_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    char *d_encrypted_password;
    checkCudaErrors(hipMalloc((void **) &d_encrypted_password, pwd_mem_size));
    //copy input to device
    checkCudaErrors(hipMemcpy(d_encrypted_password, encrypted_password, pwd_mem_size, hipMemcpyHostToDevice));

    // setup execution parameters
    const unsigned long numberIterations = pow(2,24);
    const uint num_threads = pow(2,10) / key_list_size;
    uint num_blocks = 1;
    const unsigned long max_num_threads = pow(2,33);
    while (search_space_size > num_blocks * num_threads * numberIterations
        && num_blocks * num_threads * numberIterations < max_num_threads) {
        num_blocks++;
    }
    printf("Launching %lu iterations...\n", numberIterations);
    printf("Launching %d blocks per iteration...\n", num_blocks);
    printf("Launching %d threads per block...\n", num_threads * key_list_size);
    printf("Launching %d threads per iteration...\n", num_blocks * num_threads * key_list_size);
    printf("Launching %lu total threads...\n", num_blocks * num_threads * key_list_size * numberIterations);

    dim3 grid(num_blocks, 1, 1);
    dim3 threads(num_threads, key_list_size, 1);

    // allocate mem for the result on host side
    char *decrypted_password = (char *) malloc(pwd_mem_size);

    // execute the kernel
    float totalTime = 0;
    for (uint i=0; i < numberIterations; i++) {
        hipEventRecord(start);
        crackPassword<<<grid, threads>>>(d_encrypted_password, numberIterations, i);
        hipEventRecord(stop);

        // check if kernel execution generated an error
        getLastCudaError("Kernel execution failed");
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // copy result from device to host
        typeof(d_answer) answer;
        checkCudaErrors(hipMemcpyFromSymbol(&answer, HIP_SYMBOL(d_answer), sizeof(answer), 0, hipMemcpyDeviceToHost));
        
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        totalTime += milliseconds;

        if (answer != 0) {            
            ulong_to_char_array(answer, decrypted_password);
            printf("Decrypted password: %s \n", decrypted_password);
            break;
        }
    }
    printf("Processing time: %f (ms)\n", totalTime);

    // cleanup memory
    free(decrypted_password);
    checkCudaErrors(hipFree(d_encrypted_password));

    exit(EXIT_SUCCESS);
}
