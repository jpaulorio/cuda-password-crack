#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/stat.h>
#include <dirent.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

__device__ int d_strcmp (char *s1, char *s2) {
    char *tmp_s1 = s1;
    char *tmp_s2 = s2;
    for(; *tmp_s1 == *tmp_s2; ++tmp_s1, ++tmp_s2) {
        if(*tmp_s1 == 0)
            return 0;
    }
    return *(unsigned char *)tmp_s1 < *(unsigned char *)tmp_s2 ? -1 : 1;
}

__device__ void d_strcpy (char *origin, char *destination) {
    char *tmp = origin;
    int idx = 0;
    for (; *tmp != 0; ++idx, ++tmp) {
        destination[idx] = *tmp;
    }
    destination[idx] = 0;
}

__device__ void d_encrypt(char *uncrypted, char *encryption_key, int key_length, char *encrypted) {
    char *tmp_uncrypted = uncrypted;
    char *tmp_encrypted = encrypted;
    for (uint i = 0; *tmp_uncrypted != 0; ++i, ++tmp_uncrypted, ++tmp_encrypted) {
        if (*tmp_uncrypted != 0) {
            uint key_index = i % key_length;
            *tmp_encrypted = (*tmp_uncrypted + encryption_key[key_index]) % 128;
        } else {
            *tmp_encrypted = 0;
        }
    }
}

__device__ void fill_with_zeros(char *array) {
    int array_lenght = sizeof(array);
    for (int i=0; i < array_lenght - 1; i++) {
        array[i] = 0;
    }
    array[array_lenght - 1] = 0;
}

__device__ void d_ulong_to_char_array(unsigned long search_pos, unsigned long search_space_size, char *output) {
    const uint total_no_ascii_chars = 128;
    char pwd_candidate[7];
    fill_with_zeros(pwd_candidate);

    unsigned int integer_part = search_pos / total_no_ascii_chars;
    unsigned int remainder = search_pos % total_no_ascii_chars;
    uint idx = 0;
    pwd_candidate[idx] = remainder;
    pwd_candidate[idx + 1] = integer_part;
    printf("integer: %lu\n", integer_part);
    printf("remainder: %lu\n", remainder);
    while (integer_part > 0) {
        pwd_candidate[idx] = remainder;
        pwd_candidate[idx + 1] = integer_part;
        integer_part = integer_part / total_no_ascii_chars;
        remainder = integer_part % total_no_ascii_chars;
        printf("integer: %lu\n", integer_part);
        printf("remainder: %lu\n", remainder);
        idx++;
    }

    printf("candidate: %s\n", pwd_candidate);

    d_strcpy(pwd_candidate, output);
}

__global__ void
crackPassword(
    int g_encrypted_password_length, char *g_encrypted_password, char *g_decrypted_password,
    int g_encryption_key_length, char *g_encryption_key,
    unsigned long g_search_space_size, int g_found)
{
    __shared__ char s_encrypted_password[7];
    __shared__ char s_encryption_key[4];
    char temp_password[7];
    char temp_encrypted_password[7];
    
    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int num_threads = blockDim.x;
    const unsigned long l_search_space_size = g_search_space_size;
    const unsigned long start_search = 0;
    const unsigned long end_search = l_search_space_size;
    const int key_length = g_encryption_key_length;
    unsigned long search_pos = start_search;
    unsigned int i_found = g_found;

    if (tid == 0) {
        d_strcpy(g_encrypted_password, s_encrypted_password);
        d_strcpy(g_encryption_key, s_encryption_key);
    }
    __syncthreads();

    while (!g_found) {
        d_ulong_to_char_array(search_pos, l_search_space_size, temp_password);
        printf("temp pwd: %s\n", temp_password);
        d_encrypt(temp_password, s_encryption_key, key_length, temp_encrypted_password);
        printf("temp enc pwd: %s\n", temp_encrypted_password);
        i_found = d_strcmp(temp_encrypted_password, s_encrypted_password) == 0;
        printf("i found: %d\n", i_found);
        if (i_found) {
            g_found = 1;
        }
        search_pos++;
    }

    if (i_found) {
        printf("found!!!\n");
        d_strcpy(temp_password, g_decrypted_password);
    }
}

int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    
    unsigned int num_threads = 1;
    unsigned int pwd_max_size = 32 + 1;
    unsigned int key_max_size = 32 + 1;
    
    char encrypted_password[pwd_max_size];
    char encryption_key[key_max_size];
    printf("Enter the encrypted password:\n");
    scanf("%32s", encrypted_password);
    printf("Enter the encryption key:\n");
    scanf("%32s", encryption_key);
    
    uint pwd_size = strlen(encrypted_password);
    uint key_size = strlen(encryption_key);
    
    unsigned int pwd_mem_size = (pwd_size + 1) * sizeof(char);
    unsigned int key_mem_size = (key_size + 1) * sizeof(char);
    unsigned long search_space_size = pow(128, pwd_size);
    printf("Search space size: %lu\n", search_space_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    char *d_encrypted_password, *d_decrypted_password, *d_encryption_key;
    checkCudaErrors(hipMalloc((void **) &d_encrypted_password, pwd_mem_size));
    checkCudaErrors(hipMalloc((void **) &d_encryption_key, key_mem_size));
    //output
    checkCudaErrors(hipMalloc((void **) &d_decrypted_password, pwd_mem_size));
    
    checkCudaErrors(hipMemcpy(d_encrypted_password, encrypted_password, pwd_mem_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_encryption_key, encryption_key, key_mem_size, hipMemcpyHostToDevice));

    // setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);

    hipEventRecord(start);
    // execute the kernel
    crackPassword<<<grid, threads>>>(pwd_size, d_encrypted_password, d_decrypted_password, key_size, d_encryption_key, search_space_size, 0);
    hipEventRecord(stop);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    char *decrypted_password = (char *) malloc(pwd_mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(decrypted_password, d_decrypted_password, pwd_mem_size, hipMemcpyDeviceToHost));

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Decrypted password: %s \n", decrypted_password);

    printf("Processing time: %f (ms)\n", milliseconds);

    // cleanup memory
    free(decrypted_password);
    checkCudaErrors(hipFree(d_encrypted_password));
    checkCudaErrors(hipFree(d_encryption_key));
    checkCudaErrors(hipFree(d_decrypted_password));

    exit(EXIT_SUCCESS);
}
