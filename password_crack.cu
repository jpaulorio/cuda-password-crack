#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/stat.h>
#include <dirent.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define total_no_ascii_chars 95

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

__device__ void fill_with_zeros(char *array, uint array_lenght) {
    for (int i=0; i < array_lenght; i++) {
        array[i] = 0;
    }
}

__device__ int d_strcmp (char *s1, char *s2) {
    for(int i=0; i < 7; i++) {
        if(s1[i] != s2[i])
            return 1;
    }
    return 0;
}

__device__ void d_strcpy (char *origin, char *destination) {
    for (int i=0; i < 7; i++) {
        destination[i] = origin[i];
    }
}

__device__ void d_ulong_to_char_array(unsigned long search_pos, char *output) {
    char pwd_candidate[256];
    fill_with_zeros(pwd_candidate, 256);

    unsigned long integer_part = search_pos / total_no_ascii_chars;
    unsigned long remainder = search_pos % total_no_ascii_chars;
    uint idx = 0;
    pwd_candidate[idx] = remainder + 32;
    pwd_candidate[idx + 1] = integer_part + 32;

    while (integer_part > 0) {
        idx++;
        remainder = integer_part % total_no_ascii_chars;
        integer_part = integer_part / total_no_ascii_chars;
        pwd_candidate[idx] = remainder + 32;
        pwd_candidate[idx + 1] = integer_part + 32;
    }
    pwd_candidate[idx + 1] = 0;

    d_strcpy(pwd_candidate, output);
}

__device__ void d_encrypt(uint input, uint encryption_key, char *encrypted) {
    fill_with_zeros(encrypted, 256);
    ulong tmp_pwd = input * encryption_key;
    d_ulong_to_char_array(tmp_pwd, encrypted);
}

__device__ int g_found = 0;

__global__ void
crackPassword(
    int g_encrypted_password_length, char *g_encrypted_password, char *g_decrypted_password,
    unsigned long g_search_space_size)
{
    __shared__ char s_encrypted_password[7];

    char temp_password[7];
    char temp_encrypted_password[256];
    
    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int num_threads = blockDim.x;
    const unsigned int global_tid = bid * num_threads + tid;
    const unsigned int global_num_threads = gridDim.x * blockDim.x;
    uint key_list_size = 90;
    const uint encryption_keys[] = {
        31, 37, 41, 43, 47, 53, 59, 61, 67, 71,
        73, 79, 83, 89, 97, 101, 103, 107, 109, 113,
        127, 131, 137, 139, 149, 151, 157, 163, 167, 173,
        179, 181, 191, 193, 197, 199, 211, 223, 227, 229,
        233, 239, 241, 251, 257, 263, 269, 271, 277, 281,
        283, 293, 307, 311, 313, 317, 331, 337, 347, 349,
        353, 359, 367, 373, 379, 383, 389, 397, 401, 409,
        419, 421, 431, 433, 439, 443, 449, 457, 461, 463, 
        467, 479, 487, 491, 499, 503, 509, 521, 523, 541
    };

    const unsigned long l_search_space_size = g_search_space_size;
    unsigned long chunk_size = l_search_space_size / global_num_threads;

    if (chunk_size == 0) {
        chunk_size = l_search_space_size / num_threads;

        if (chunk_size == 0) {
            chunk_size = l_search_space_size;
        }
    }

    const unsigned long start_search = global_tid * chunk_size;
    unsigned long end_search = start_search + chunk_size;
    if (start_search >= l_search_space_size) {
        return;
    }
    if (end_search > l_search_space_size) {
        end_search = l_search_space_size;
    }
    unsigned long search_pos = start_search;

    if (tid == 0) {
        fill_with_zeros(s_encrypted_password, 7);
        d_strcpy(g_encrypted_password, s_encrypted_password);

        if (bid == 0) {
            printf("Global num threads: %d\n", global_num_threads);
            printf("Chunk size: %lu\n", chunk_size);
        }
    }
    __syncthreads();

    while (!g_found && search_pos < end_search) {
        uint key_search_pos = 0;        

        while (!g_found && key_search_pos < key_list_size) {
            uint key = encryption_keys[key_search_pos];

            d_encrypt(search_pos, key, temp_encrypted_password);

            if (d_strcmp(temp_encrypted_password, s_encrypted_password) == 0) {
                d_ulong_to_char_array(search_pos, temp_password);
                d_strcpy(temp_password, g_decrypted_password);
                printf("Password was found by thread %d!\nDetails: start|end|current - %lu:%lu:%lu\n",
                    global_tid, start_search, end_search, search_pos);
                g_found = 1;
            }
            key_search_pos++;
        }
        search_pos++;
    }
}

int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

void
runTest(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    unsigned int pwd_max_size = 32 + 1;
    unsigned int key_max_size = 32 + 1;
    
    char encrypted_password[pwd_max_size];
    char encryption_key[key_max_size];
    printf("Enter the encrypted password:\n");
    scanf("%6s", encrypted_password);
    
    uint pwd_size = strlen(encrypted_password);
    uint key_size = strlen(encryption_key);
    
    unsigned int pwd_mem_size = (pwd_size + 1) * sizeof(char);
    unsigned long search_space_size = pow(total_no_ascii_chars, 5);
    printf("Search space size: %lu\n", search_space_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    char *d_encrypted_password, *d_decrypted_password;
    checkCudaErrors(hipMalloc((void **) &d_encrypted_password, pwd_mem_size));
    //output
    checkCudaErrors(hipMalloc((void **) &d_decrypted_password, pwd_mem_size));
    
    checkCudaErrors(hipMemcpy(d_encrypted_password, encrypted_password, pwd_mem_size, hipMemcpyHostToDevice));
    hipStreamQuery(0);

    // setup execution parameters
    unsigned int num_threads = 512;
    unsigned int num_blocks = 1;
    unsigned long max_num_threads = pow(2,21);
    while (search_space_size > num_blocks * num_threads && num_blocks * num_threads < max_num_threads) {
        num_blocks++;
    }
    printf("Launching %d threads...\n", num_blocks * num_threads);
    // unsigned int num_blocks = pow(2,21);
    dim3  grid(num_blocks, 1, 1);
    dim3  threads(num_threads, 1, 1);

    hipEventRecord(start);
    // execute the kernel
    crackPassword<<<grid, threads>>>(pwd_size, d_encrypted_password, d_decrypted_password, search_space_size);
    hipStreamQuery(0);
    hipEventRecord(stop);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // allocate mem for the result on host side
    char *decrypted_password = (char *) malloc(pwd_mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(decrypted_password, d_decrypted_password, pwd_mem_size, hipMemcpyDeviceToHost));
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Decrypted password: %s \n", decrypted_password);

    printf("Processing time: %f (ms)\n", milliseconds);

    // cleanup memory
    free(decrypted_password);
    checkCudaErrors(hipFree(d_encrypted_password));
    checkCudaErrors(hipFree(d_decrypted_password));

    exit(EXIT_SUCCESS);
}
