#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/stat.h>
#include <dirent.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

__device__ int d_strcmp (const char *s1, const char *s2) {
    for(; *s1 == *s2; ++s1, ++s2) {
        if(*s1 == 0)
            return 0;
    }
    return *(unsigned char *)s1 < *(unsigned char *)s2 ? -1 : 1;
}

__device__ void d_strcpy (char *origin, char *destination) {
    char *tmp = origin;
    int idx = 0;
    for (; *tmp != 0; ++idx, ++tmp) {
        destination[idx] = *tmp;
    }
    destination[idx] = 0;
}

__device__ void d_encrypt(char *uncrypted, char *encryption_key, int key_length, char *encrypted) {
    for (uint i = 0; *uncrypted != 0; ++i, ++uncrypted, ++encrypted) {
        printf("here2 %c\n", *uncrypted);
        if (*uncrypted != 0) {
            uint key_index = i % key_length;
            *encrypted = (*uncrypted + encryption_key[key_index]) % 128;
        } else {
            *encrypted = 0;
        }
    }
}

__global__ void
crackPassword(
    int g_encrypted_password_length, char *g_encrypted_password, char *g_decrypted_password,
    int g_encryption_key_length, char *g_encryption_key,
    int g_found)
{
    __shared__ char s_encrypted_password[7];
    __shared__ char s_encryption_key[4];
    char temp_password[7];
    
    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x;
    const unsigned int num_threads = blockDim.x;
    unsigned int i_found = g_found;

    if (tid == 0) {
        d_strcpy(g_encrypted_password, s_encrypted_password);
        d_strcpy(g_encryption_key, s_encryption_key);
    }
    __syncthreads();

    while (!g_found) {
        d_strcpy(s_encrypted_password, temp_password);
        i_found = 1;
        g_found = 1;
    }

    if (i_found) {
        printf("found!!!\n");
        d_strcpy(temp_password, g_decrypted_password);
    }
}

int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    unsigned int num_threads = 1;
    unsigned int pwd_max_size = 32 + 1;
    unsigned int key_max_size = 32 + 1;
    
    char encrypted_password[pwd_max_size];
    char encryption_key[key_max_size];
    printf("Enter the encrypted password:\n");
    scanf("%32s", encrypted_password);
    printf("Enter the encryption key:\n");
    scanf("%32s", encryption_key);
    
    uint pwd_size = strlen(encrypted_password) + 1;
    uint key_size = strlen(encryption_key) + 1;

    unsigned int pwd_mem_size = pwd_size * sizeof(char);
    unsigned int key_mem_size = key_size * sizeof(char);

    char *d_encrypted_password, *d_decrypted_password, *d_encryption_key;
    checkCudaErrors(hipMalloc((void **) &d_encrypted_password, pwd_mem_size));
    checkCudaErrors(hipMalloc((void **) &d_encryption_key, key_mem_size));
    //output
    checkCudaErrors(hipMalloc((void **) &d_decrypted_password, pwd_mem_size));
    
    checkCudaErrors(hipMemcpy(d_encrypted_password, encrypted_password, pwd_mem_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_encryption_key, encryption_key, key_mem_size, hipMemcpyHostToDevice));

    // setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel
    crackPassword<<<grid, threads>>>(pwd_size, d_encrypted_password, d_decrypted_password, key_size, d_encryption_key, 0);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    char *decrypted_password = (char *) malloc(pwd_mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(decrypted_password, d_decrypted_password, pwd_mem_size, hipMemcpyDeviceToHost));

    printf("Decrypted password: %s \n", decrypted_password);

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    // cleanup memory
    free(encrypted_password);
    free(encryption_key);
    free(decrypted_password);
    checkCudaErrors(hipFree(d_encrypted_password));
    checkCudaErrors(hipFree(d_encryption_key));
    checkCudaErrors(hipFree(d_decrypted_password));

    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
